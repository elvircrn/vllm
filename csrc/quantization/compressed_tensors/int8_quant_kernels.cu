#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>

#include <cmath>

#include "../../dispatch_utils.h"
#include "../vectorization_utils.cuh"

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
  #include <cub/util_type.cuh>
#else
  #include <hipcub/hipcub.hpp>
  #include <hipcub/util_type.hpp>
#endif

static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static constexpr auto i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());

  // To match the rounding mode of CUDA, we use nearbyint.
  // It uses the current rounding mode, which is always FE_TONEAREST on HIP.
  // If that changes in the future, we may need to set the rounding mode
  // explicitly, either at runtime or compile time.
  float dst = std::nearbyint(x);

  // saturate

  // See https://github.com/pytorch/pytorch/issues/127666
  // See https://github.com/llvm/llvm-project/issues/95183
  // hip-clang std::clamp __glibcxx_assert_fail host function when building on
  // Arch/gcc14. The following replaces std::clamp usage with similar logic
  // dst = std::clamp(dst, i8_min, i8_max);
  dst = (dst < i8_min) ? i8_min : (dst > i8_max) ? i8_max : dst;
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

static inline __device__ int32_t float_to_int32_rn(float x) {
#ifdef USE_ROCM
  // int32_max is not exactly representable as float.
  // Therefore, we need to be careful and manually return int32_max on overflow.
  // For symmetry, we also do the same for int32_min, even though it is exactly
  // representable as float and the conversion should be exact.
  static constexpr auto i32_min = std::numeric_limits<int32_t>::min();
  static constexpr auto i32_min_f = static_cast<float>(i32_min);
  static constexpr auto i32_max = std::numeric_limits<int32_t>::max();
  static constexpr auto i32_max_f = static_cast<float>(i32_max);

  // To match the rounding mode of CUDA, we use nearbyint.
  // It uses the current rounding mode, which is always FE_TONEAREST on HIP.
  // If that changes in the future, we may need to set the rounding mode
  // explicitly, either at runtime or compile time.
  float dst = std::nearbyint(x);

  // saturate on the higher end.
  if (dst >= i32_max_f) {
    return i32_max;
  }
  // saturate on the lower end.
  if (dst <= i32_min_f) {
    return i32_min;
  }

  return static_cast<int32_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s32.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int32_t&>(dst);
#endif
}

static inline __device__ int8_t int32_to_int8(int32_t x) {
#ifdef USE_ROCM
  static constexpr auto i8_min =
      static_cast<int32_t>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<int32_t>(std::numeric_limits<int8_t>::max());

  // saturate

  // See https://github.com/pytorch/pytorch/issues/127666
  // See https://github.com/llvm/llvm-project/issues/95183
  // hip-clang std::clamp __glibcxx_assert_fail host function when building on
  // Arch/gcc14. The following replaces std::clamp usage with similar logic
  // int32_t dst = std::clamp(x, i8_min, i8_max);
  int32_t dst = (x < i8_min) ? i8_min : (x > i8_max) ? i8_max : x;
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.sat.s8.s32 %0, %1;" : "=r"(dst) : "r"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_t>
__global__ void static_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    const scale_t* scale_ptr, const int hidden_size) {
  int tid = threadIdx.x;
  int stride = blockDim.x;
  float scale = *scale_ptr;

  const scalar_t* row_in = input + blockIdx.x * hidden_size;
  int8_t* row_out = output + blockIdx.x * hidden_size;

  vectorize_with_alignment(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(vec_n_t<int8_t, 16> & dst,
                     const vec_n_t<scalar_t, 16>& src) {
#pragma unroll
        for (int k = 0; k < 16; ++k)
          dst.val[k] = float_to_int8_rn(float(src.val[k]) / scale);
      },
      [=] __device__(int8_t& dst, const scalar_t& src) {
        dst = float_to_int8_rn(float(src) / scale);
      });
}

template <typename scalar_t, typename scale_t, typename azp_t>
__global__ void static_scaled_int8_azp_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    const scale_t* scale_ptr, const azp_t* azp_ptr, const int hidden_size) {
  int tid = threadIdx.x;
  int stride = blockDim.x;
  float scale = *scale_ptr;
  azp_t azp = *azp_ptr;
  float inv_s = 1.0f / scale;

  const scalar_t* row_in = input + blockIdx.x * hidden_size;
  int8_t* row_out = output + blockIdx.x * hidden_size;

  vectorize_with_alignment(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(vec_n_t<int8_t, 16> & dst,
                     const vec_n_t<scalar_t, 16>& src) {
#pragma unroll
        for (int k = 0; k < 16; ++k) {
          float v = float(src.val[k]) * inv_s;
          dst.val[k] = int32_to_int8(float_to_int32_rn(v) + azp);
        }
      },
      [=] __device__(int8_t& dst, const scalar_t& src) {
        float v = float(src) * inv_s;
        dst = int32_to_int8(float_to_int32_rn(v) + azp);
      });
}

template <typename scalar_t, typename scale_t>
__global__ void dynamic_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    scale_t* scale_out, const int hidden_size) {
  int tid = threadIdx.x;
  int stride = blockDim.x;

  const scalar_t* row_in = input + blockIdx.x * hidden_size;
  int8_t* row_out = output + blockIdx.x * hidden_size;

  // calculate for absmax
  float thread_max = 0.f;
  for (int i = tid; i < hidden_size; i += stride) {
    float v = fabsf(float(row_in[i]));
    thread_max = fmaxf(thread_max, v);
  }
  using BlockReduce = hipcub::BlockReduce<float, 256>;
  __shared__ typename BlockReduce::TempStorage tmp;
  float block_max = BlockReduce(tmp).Reduce(thread_max, hipcub::Max{}, blockDim.x);
  __shared__ float absmax;
  if (tid == 0) {
    absmax = block_max;
    scale_out[blockIdx.x] = absmax / 127.f;
  }
  __syncthreads();

  float inv_s = (absmax == 0.f) ? 0.f : 127.f / absmax;

  // 2. quantize
  vectorize_with_alignment(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(vec_n_t<int8_t, 16> & dst,
                     const vec_n_t<scalar_t, 16>& src) {
#pragma unroll
        for (int k = 0; k < 16; ++k)
          dst.val[k] = float_to_int8_rn(float(src.val[k]) * inv_s);
      },
      [=] __device__(int8_t& dst, const scalar_t& src) {
        dst = float_to_int8_rn(float(src) * inv_s);
      });
}

template <typename scalar_t, typename scale_t, typename azp_t>
__global__ void dynamic_scaled_int8_azp_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    scale_t* scale_out, azp_t* azp_out, const int hidden_size) {
  int tid = threadIdx.x;
  int stride = blockDim.x;

  const scalar_t* row_in = input + blockIdx.x * hidden_size;
  int8_t* row_out = output + blockIdx.x * hidden_size;

  // 1. calculate min & max
  float tmin = FLT_MAX;
  float tmax = -FLT_MAX;
  for (int i = tid; i < hidden_size; i += stride) {
    float v = float(row_in[i]);
    tmin = fminf(tmin, v);
    tmax = fmaxf(tmax, v);
  }

  // This is used for BlockReduce
  struct MinMax {
    float mn, mx;
  };
  struct ReduceMinMax {
    __device__ MinMax operator()(const MinMax& a, const MinMax& b) const {
      return {fminf(a.mn, b.mn), fmaxf(a.mx, b.mx)};
    }
  };

  using BlockReduce = hipcub::BlockReduce<MinMax, 256>;
  __shared__ typename BlockReduce::TempStorage tmp;

  MinMax thread_mm{tmin, tmax};
  MinMax mm = BlockReduce(tmp).Reduce(thread_mm, ReduceMinMax(), blockDim.x);
  __shared__ float scale_sh;
  __shared__ azp_t azp_sh;
  if (tid == 0) {
    float s = (mm.mx - mm.mn) / 255.f;
    float zp = nearbyintf(-128.f - mm.mn / s);  // round-to-even
    scale_sh = s;
    azp_sh = azp_t(zp);
    scale_out[blockIdx.x] = s;
    azp_out[blockIdx.x] = azp_sh;
  }
  __syncthreads();

  float inv_s = 1.f / scale_sh;
  azp_t azp = azp_sh;

  // 2. quantize
  vectorize_with_alignment(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(vec_n_t<int8_t, 16> & dst,
                     const vec_n_t<scalar_t, 16>& src) {
#pragma unroll
        for (int k = 0; k < 16; ++k) {
          float v = float(src.val[k]) * inv_s;
          dst.val[k] = int32_to_int8(float_to_int32_rn(v) + azp);
        }
      },
      [=] __device__(int8_t& dst, const scalar_t& src) {
        float v = float(src) * inv_s;
        dst = int32_to_int8(float_to_int32_rn(v) + azp);
      });
}

}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale,
                              std::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);
  TORCH_CHECK(!azp || azp->numel() == 1);

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 256));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::static_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), hidden_size);
        } else {
          vllm::static_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}

void dynamic_scaled_int8_quant(
    torch::Tensor& out,          // [..., hidden_size]
    torch::Tensor const& input,  // [..., hidden_size]
    torch::Tensor& scales, std::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scales.is_contiguous());
  TORCH_CHECK(!azp || azp->is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 256));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), hidden_size);
        } else {
          vllm::dynamic_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}
