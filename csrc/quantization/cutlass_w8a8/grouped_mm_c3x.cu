#include "hip/hip_runtime.h"
#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#include "cutlass/cutlass.h"
#include "grouped_mm_c3x.cuh"

#include "grouped_mm_c3x.cuh"

using namespace cute;

#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 900
  #define ENABLE_SM90_KERNEL_LEVEL 1
#endif

namespace {

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_fp8_config_default {
  // M in (16, inf)
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedPingpong;
  using TileShape = cute::Shape<cute::_64, cute::_256, cute::_128>;
  using ClusterShape = cute::Shape<cute::_1, cute::_2, cute::_1>;

  using Cutlass3xGemm =
      cutlass_3x_group_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                            KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_fp8_config_M16 {
  // M in [1, 16]
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedPingpong;
  using TileShape = cute::Shape<cute::_64, cute::_64, cute::_128>;
  using ClusterShape = cute::Shape<cute::_1, cute::_4, cute::_1>;

  using Cutlass3xGemm =
      cutlass_3x_group_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                            KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_fp8_config_K8192 {
  // K in [8192, inf)
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedPingpong;
  using TileShape = cute::Shape<cute::_128, cute::_128, cute::_128>;
  using ClusterShape = cute::Shape<cute::_1, cute::_8, cute::_1>;

  using Cutlass3xGemm =
      cutlass_3x_group_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                            KernelSchedule, EpilogueSchedule>;
};

template <typename InType, typename OutType,
          template <typename, typename, typename> typename Epilogue>
struct sm90_fp8_config_N8192 {
  // N in [8192, inf)
  static_assert(std::is_same<InType, cutlass::float_e4m3_t>());
  using KernelSchedule =
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpongFP8FastAccum;
  using EpilogueSchedule =
      cutlass::epilogue::PtrArrayTmaWarpSpecializedPingpong;
  using TileShape = cute::Shape<cute::_64, cute::_128, cute::_256>;
  using ClusterShape = cute::Shape<cute::_1, cute::_8, cute::_1>;

  using Cutlass3xGemm =
      cutlass_3x_group_gemm<InType, OutType, Epilogue, TileShape, ClusterShape,
                            KernelSchedule, EpilogueSchedule>;
};

}  // namespace

void cutlass_grouped_mm_sm90(
    torch::Tensor& out_tensors, torch::Tensor const& a_tensors,
    torch::Tensor const& b_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& expert_offsets,
    torch::Tensor const& problem_sizes, torch::Tensor const& a_strides,
    torch::Tensor const& b_strides, torch::Tensor const& c_strides) {
  TORCH_CHECK(a_tensors.size(0) > 0, "No input A tensors provided.");
  TORCH_CHECK(b_tensors.size(0) > 0, "No input B tensors provided.");
  TORCH_CHECK(out_tensors.size(0) > 0, "No output tensors provided.");

  TORCH_CHECK(a_tensors.dtype() == torch::kFloat8_e4m3fn,
              "A tensors must be of type float8_e4m3fn.");
  TORCH_CHECK(b_tensors.dtype() == torch::kFloat8_e4m3fn,
              "B tensors must be of type float8_e4m3fn.");

  using Cutlass3xGemmN8192 = typename sm90_fp8_config_N8192<
      ElementAB_Type, ElementC_Type,
      vllm::c3x::ScaledEpilogueArray>::Cutlass3xGemm;
  using Cutlass3xGemmK8192 = typename sm90_fp8_config_K8192<
      ElementAB_Type, ElementC_Type,
      vllm::c3x::ScaledEpilogueArray>::Cutlass3xGemm;
  using Cutlass3xGemmM16 = typename sm90_fp8_config_M16<
      ElementAB_Type, ElementC_Type,
      vllm::c3x::ScaledEpilogueArray>::Cutlass3xGemm;
  using Cutlass3xGemmDefault = typename sm90_fp8_config_default<
      ElementAB_Type, ElementC_Type,
      vllm::c3x::ScaledEpilogueArray>::Cutlass3xGemm;

  uint32_t const m = a_tensors.size(0);
  uint32_t const n = out_tensors.size(1);
  uint32_t const k = a_tensors.size(1);

  if (n >= 8192) {
    cutlass_group_gemm_caller<Cutlass3xGemmN8192>(
        out_tensors, a_tensors, b_tensors, a_scales, b_scales, expert_offsets,
        problem_sizes, a_strides, b_strides, c_strides);
  } else if (k >= 8192) {
    cutlass_group_gemm_caller<Cutlass3xGemmK8192>(
        out_tensors, a_tensors, b_tensors, a_scales, b_scales, expert_offsets,
        problem_sizes, a_strides, b_strides, c_strides);
  } else if (m <= 16) {
    cutlass_group_gemm_caller<Cutlass3xGemmM16>(
        out_tensors, a_tensors, b_tensors, a_scales, b_scales, expert_offsets,
        problem_sizes, a_strides, b_strides, c_strides);
  } else {
    cutlass_group_gemm_caller<Cutlass3xGemmDefault>(
        out_tensors, a_tensors, b_tensors, a_scales, b_scales, expert_offsets,
        problem_sizes, a_strides, b_strides, c_strides);
  }
}

// basic correctness, currently unused, run with <<<1, num_experts>>>
__global__ void get_grouped_mm_data_kernel(
    const int* __restrict__ topk_ids, int32_t* expert_offsets,
    int32_t* problem_sizes1, int32_t* problem_sizes2, int32_t* arg_sort,
    int32_t* arg_sort_prim, int topk_length, int n, int k, int topk) {
  int expert_id = threadIdx.x;
  int num_experts = blockDim.x;

  int occurrences = 0;
  for (int i = 0; i < topk_length; ++i) {
    occurrences += (topk_ids[i] == expert_id);
  }
  problem_sizes1[expert_id * 3] = occurrences;
  problem_sizes1[expert_id * 3 + 1] = 2 * n;
  problem_sizes1[expert_id * 3 + 2] = k;
  problem_sizes2[expert_id * 3] = occurrences;
  problem_sizes2[expert_id * 3 + 1] = k;
  problem_sizes2[expert_id * 3 + 2] = n;
  __syncthreads();

  if (threadIdx.x == 0) {
    int32_t tot_offset = 0;
    expert_offsets[0] = 0;
    for (int i = 0; i < num_experts; ++i) {
      tot_offset += problem_sizes1[i * 3];
      expert_offsets[i + 1] = tot_offset;
    }
  }

  __syncthreads();

  int start = expert_offsets[expert_id];
  int end = expert_offsets[expert_id + 1];
  for (int i = 0; i < topk_length; ++i) {
    if (topk_ids[i] == expert_id) {
      arg_sort[start] = i / topk;
      arg_sort_prim[i] = start;
      ++start;
      if (start == end) {
        break;
      }
    }
  }
}

constexpr int THREADS_PER_EXPERT = 512;

__global__ void compute_problem_sizes(const int* __restrict__ topk_ids,
                                      int32_t* problem_sizes1,
                                      int32_t* problem_sizes2,
                                      int32_t* atomic_buffer,
                                      const int topk_length, const int n,
                                      const int k) {
  int expert_id = blockIdx.x;

  int occurrences = 0;
  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    occurrences += (topk_ids[i] == expert_id);
  }
  atomicAdd(&atomic_buffer[expert_id], occurrences);
  __syncthreads();

  if (threadIdx.x == 0) {
    int final_occurrences = atomic_buffer[expert_id];
    problem_sizes1[expert_id * 3] = final_occurrences;
    problem_sizes1[expert_id * 3 + 1] = 2 * n;
    problem_sizes1[expert_id * 3 + 2] = k;
    problem_sizes2[expert_id * 3] = final_occurrences;
    problem_sizes2[expert_id * 3 + 1] = k;
    problem_sizes2[expert_id * 3 + 2] = n;
  }
}

__global__ void compute_expert_offsets(
    const int32_t* __restrict__ problem_sizes1, int32_t* expert_offsets,
    int32_t* atomic_buffer, const int num_experts) {
  int32_t tot_offset = 0;
  expert_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    tot_offset += problem_sizes1[i * 3];
    expert_offsets[i + 1] = tot_offset;
  }
}

__global__ void compute_arg_sorts(const int* __restrict__ topk_ids,
                                  int32_t* arg_sort, int32_t* arg_sort_prim,
                                  int32_t* atomic_buffer, const int topk_length,
                                  const int topk) {
  int expert_id = blockIdx.x;

  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    if (topk_ids[i] == expert_id) {
      int start = atomicAdd(&atomic_buffer[expert_id], 1);
      arg_sort[start] = i / topk;
      arg_sort_prim[i] = start;
    }
  }
}

constexpr int THREADS_PER_EXPERT_2 = 32;

// 1 warp per expert
// 4 experts per block
__global__ void compute_problem_sizes_multi_expert(
    const int* __restrict__ topk_ids, int32_t* problem_sizes1,
    int32_t* problem_sizes2, int32_t* atomic_buffer, const int topk_length,
    const int n, const int k) {
  int expert_id = blockIdx.x * 4 + threadIdx.x / THREADS_PER_EXPERT_2;
  int start = threadIdx.x % THREADS_PER_EXPERT_2;

  int occurrences = 0;
  for (int i = start; i < topk_length; i += THREADS_PER_EXPERT_2) {
    occurrences += (topk_ids[i] == expert_id);
  }
  atomicAdd(&atomic_buffer[expert_id], occurrences);
  // we only need this if #threads/expert > warp_size
  if constexpr (THREADS_PER_EXPERT_2 > 32) {
    __syncthreads();
  }

  if (start == 0) {
    int final_occurrences = atomic_buffer[expert_id];
    problem_sizes1[expert_id * 3] = final_occurrences;
    problem_sizes1[expert_id * 3 + 1] = 2 * n;
    problem_sizes1[expert_id * 3 + 2] = k;
    problem_sizes2[expert_id * 3] = final_occurrences;
    problem_sizes2[expert_id * 3 + 1] = k;
    problem_sizes2[expert_id * 3 + 2] = n;
  }
}

__global__ void compute_arg_sorts_multi_expert(
    const int* __restrict__ topk_ids, int32_t* arg_sort, int32_t* arg_sort_prim,
    int32_t* atomic_buffer, const int topk_length, const int topk) {
  int expert_id = blockIdx.x * 4 + threadIdx.x / THREADS_PER_EXPERT_2;
  int start = threadIdx.x % THREADS_PER_EXPERT_2;

  for (int i = start; i < topk_length; i += THREADS_PER_EXPERT_2) {
    if (topk_ids[i] == expert_id) {
      int start = atomicAdd(&atomic_buffer[expert_id], 1);
      arg_sort[start] = i / topk;
      arg_sort_prim[i] = start;
    }
  }
}

void get_grouped_mm_data_caller(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& arg_sort, torch::Tensor& arg_sort_prim,
    const int64_t num_experts, const int64_t n, const int64_t k) {
  auto stream = at::cuda::getCurrentCUDAStream(topk_ids.device().index());
  auto options_int32 =
      torch::TensorOptions().dtype(torch::kInt32).device(topk_ids.device());
  torch::Tensor atomic_buffer = torch::zeros(num_experts, options_int32);

  // TODO this is an alternative way to block kernels
  constexpr bool multi_expert_blocks = false;
  if constexpr (multi_expert_blocks) {
    int num_blocks = (num_experts + 3) / 4;
    int num_threads = THREADS_PER_EXPERT_2 * 4;
    compute_problem_sizes_multi_expert<<<num_blocks, num_threads, 0, stream>>>(
        (const int32_t*)topk_ids.data_ptr(),
        (int32_t*)problem_sizes1.data_ptr(),
        (int32_t*)problem_sizes2.data_ptr(), (int32_t*)atomic_buffer.data_ptr(),
        topk_ids.numel(), n, k);
    compute_expert_offsets<<<1, 1, 0, stream>>>(
        (const int32_t*)problem_sizes1.data_ptr(),
        (int32_t*)expert_offsets.data_ptr(), (int32_t*)atomic_buffer.data_ptr(),
        num_experts);
    compute_arg_sorts_multi_expert<<<num_blocks, num_threads, 0, stream>>>(
        (const int32_t*)topk_ids.data_ptr(), (int32_t*)arg_sort.data_ptr(),
        (int32_t*)arg_sort_prim.data_ptr(), (int32_t*)atomic_buffer.data_ptr(),
        topk_ids.numel(), topk_ids.size(1));
    return;
  }

  int num_threads = min(THREADS_PER_EXPERT, topk_ids.numel());
  compute_problem_sizes<<<num_experts, num_threads, 0, stream>>>(
      (const int32_t*)topk_ids.data_ptr(), (int32_t*)problem_sizes1.data_ptr(),
      (int32_t*)problem_sizes2.data_ptr(), (int32_t*)atomic_buffer.data_ptr(),
      topk_ids.numel(), n, k);
  compute_expert_offsets<<<1, 1, 0, stream>>>(
      (const int32_t*)problem_sizes1.data_ptr(),
      (int32_t*)expert_offsets.data_ptr(), (int32_t*)atomic_buffer.data_ptr(),
      num_experts);
  compute_arg_sorts<<<num_experts, num_threads, 0, stream>>>(
      (const int32_t*)topk_ids.data_ptr(), (int32_t*)arg_sort.data_ptr(),
      (int32_t*)arg_sort_prim.data_ptr(), (int32_t*)atomic_buffer.data_ptr(),
      topk_ids.numel(), topk_ids.size(1));
}
