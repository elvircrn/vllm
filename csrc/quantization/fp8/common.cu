#include "hip/hip_runtime.h"
#include "common.cuh"
#include "dispatch_utils.h"
#include "../vectorization_utils.cuh"
#include <c10/cuda/CUDAGuard.h>

#ifndef USE_ROCM
  #include <hipcub/hipcub.hpp>
#else
  #include <hipcub/hipcub.hpp>
#endif

namespace vllm {

template <typename scalar_t, typename fp8_type>
__global__ void scaled_fp8_quant_kernel(fp8_type* __restrict__ out,
                                        const scalar_t* __restrict__ input,
                                        const float* __restrict__ scale,
                                        int64_t num_elems) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  // Invert the scale so that we can use multiplications to avoid expensive
  // division.
  const float inverted_scale = 1.0f / (*scale);
  vectorize_with_alignment<16>(
      input, out, num_elems, tid, blockDim.x * gridDim.x,
      [=] __device__(fp8_type & dst, const scalar_t& src) {
        dst = scaled_fp8_conversion<true, fp8_type>(static_cast<float>(src),
                                                    inverted_scale);
      });
}

template <typename scalar_t, typename fp8_type>
__global__ void dynamic_per_token_scaled_fp8_quant_kernel(
    fp8_type* __restrict__ out, float* __restrict__ scale,
    scalar_t const* __restrict__ input, float const* __restrict__ scale_ub,
    const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;

  // Use int64 to avoid overflowing an int32 when calculating this offset
  int64_t offset = static_cast<int64_t>(token_idx) * hidden_size;
  scalar_t const* __restrict__ token_input = &input[offset];
  fp8_type* __restrict__ token_output = &out[offset];

  // 1) compute per-token absmax
  float absmax_val = 0.0f;
  vectorize_read_with_alignment<16>(token_input, hidden_size, tid, blockDim.x,
                                    [&] __device__(const scalar_t& src) {
                                      const float v =
                                          fabsf(static_cast<float>(src));
                                      absmax_val = fmaxf(absmax_val, v);
                                    });

  using BlockReduce = hipcub::BlockReduce<float, 256>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float token_scale;
  if (tid == 0) {
    if (scale_ub) {
      token_scale = fminf(block_absmax_val_maybe, *scale_ub);
    } else {
      token_scale = block_absmax_val_maybe;
    }
    // token scale computation
    token_scale = fmaxf(token_scale / quant_type_max_v<fp8_type>,
                        min_scaling_factor<fp8_type>::val());
    scale[token_idx] = token_scale;
  }
  __syncthreads();

  // 2) quantize
  // Note that we don't use inverted scales so we can match FBGemm impl.
  vectorize_with_alignment<16>(
      token_input, token_output, hidden_size, tid, blockDim.x,
      [=] __device__(fp8_type & dst, const scalar_t& src) {
        dst = scaled_fp8_conversion<false, fp8_type>(static_cast<float>(src),
                                                     token_scale);
      });
}

}  // namespace vllm

void static_scaled_fp8_quant(torch::Tensor& out,          // [..., d]
                             torch::Tensor const& input,  // [..., d]
                             torch::Tensor const& scale)  // [1]
{
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  int const block_size = 256;
  int const num_tokens = input.numel() / input.size(-1);
  int const num_elems = input.numel();
  dim3 const grid(num_tokens);
  dim3 const block(block_size);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel_scalar_type", [&] {
        VLLM_DISPATCH_FP8_TYPES(
            out.scalar_type(), "scaled_fp8_quant_kernel_fp8_type", [&] {
              vllm::scaled_fp8_quant_kernel<scalar_t, fp8_t>
                  <<<grid, block, 0, stream>>>(
                      out.data_ptr<fp8_t>(), input.data_ptr<scalar_t>(),
                      scale.data_ptr<float>(), num_elems);
            });
      });
}

void dynamic_scaled_fp8_quant(torch::Tensor& out,          // [..., d]
                              torch::Tensor const& input,  // [..., d]
                              torch::Tensor& scale)        // [1]
{
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  int const block_size = 256;
  int const num_tokens = input.numel() / input.size(-1);
  int const num_elems = input.numel();
  dim3 const grid(num_tokens);
  dim3 const block(block_size);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "scaled_fp8_quant_kernel_scalar_type", [&] {
        VLLM_DISPATCH_FP8_TYPES(
            out.scalar_type(), "scaled_fp8_quant_kernel_fp8_type", [&] {
              vllm::segmented_max_reduction<scalar_t, fp8_t>
                  <<<grid, block, 0, stream>>>(scale.data_ptr<float>(),
                                               input.data_ptr<scalar_t>(),
                                               num_elems);
              vllm::scaled_fp8_quant_kernel<scalar_t, fp8_t>
                  <<<grid, block, 0, stream>>>(
                      out.data_ptr<fp8_t>(), input.data_ptr<scalar_t>(),
                      scale.data_ptr<float>(), num_elems);
            });
      });
}

void dynamic_per_token_scaled_fp8_quant(
    torch::Tensor& out,          // [..., d]
    torch::Tensor const& input,  // [..., d]
    torch::Tensor& scales, std::optional<at::Tensor> const& scale_ub) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  int const block_size = 256;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, block_size));

  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(),
      "dynamic_per_token_scaled_fp8_quant_kernel_scalar_type", [&] {
        VLLM_DISPATCH_FP8_TYPES(
            out.scalar_type(),
            "dynamic_per_token_scaled_fp8_quant_kernel_fp8_type", [&] {
              vllm::dynamic_per_token_scaled_fp8_quant_kernel<scalar_t, fp8_t>
                  <<<grid, block, 0, stream>>>(
                      out.data_ptr<fp8_t>(), scales.data_ptr<float>(),
                      input.data_ptr<scalar_t>(),
                      scale_ub.has_value() ? scale_ub->data_ptr<float>()
                                           : nullptr,
                      hidden_size);
            });
      });
}
