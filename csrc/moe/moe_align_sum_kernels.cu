#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <ATen/ATen.h>
#include <ATen/cuda/Atomic.cuh>

#include "../cuda_compat.h"
#include "../dispatch_utils.h"

#define CEILDIV(x, y) (((x) + (y) - 1) / (y))

namespace vllm {
namespace moe {

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad, int32_t num_experts,
    int32_t padded_num_experts, int32_t experts_per_warp, int32_t block_size,
    size_t numel, int32_t* __restrict__ cumsum) {
  extern __shared__ int32_t shared_counts[];

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int my_expert_start = warp_id * experts_per_warp;

  for (int i = 0; i < experts_per_warp; ++i) {
    if (my_expert_start + i < padded_num_experts) {
      shared_counts[warp_id * experts_per_warp + i] = 0;
    }
  }

  __syncthreads();

  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int expert_id = topk_ids[i];
    if (expert_id == -1) {
      continue;
    }
    int warp_idx = expert_id / experts_per_warp;
    int expert_offset = expert_id % experts_per_warp;
    atomicAdd(&shared_counts[warp_idx * experts_per_warp + expert_offset], 1);
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      int expert_count = 0;
      int warp_idx = (i - 1) / experts_per_warp;
      int expert_offset = (i - 1) % experts_per_warp;
      expert_count = shared_counts[warp_idx * experts_per_warp + expert_offset];

      cumsum[i] =
          cumsum[i - 1] + CEILDIV(expert_count, block_size) * block_size;
    }
    *total_tokens_post_pad = cumsum[num_experts];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }
}

template <typename scalar_t>
__global__ void count_and_sort_expert_tokens_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ cumsum_buffer,
    size_t numel) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    if (expert_id == -1) {
      continue;
    }
    int32_t rank_post_pad = atomicAdd(&cumsum_buffer[expert_id], 1);
    sorted_token_ids[rank_post_pad] = i;
  }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., topk, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    scalar_t x = 0.0;
#pragma unroll
    for (int k = 0; k < TOPK; ++k) {
      x += VLLM_LDG(&input[token_idx * TOPK * d + k * d + idx]);
    }
    out[token_idx * d + idx] = x;
  }
}

template <typename scalar_t>
__global__ void moe_align_block_size_small_batch_expert_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad, int32_t num_experts,
    int32_t block_size, size_t numel) {
  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  extern __shared__ int32_t shared_mem[];
  int32_t* cumsum = shared_mem;
  int32_t* tokens_cnts = (int32_t*)(shared_mem + num_experts + 1);

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[(threadIdx.x + 1) * num_experts + i] = 0;
  }

  for (size_t i = tid; i < numel; i += stride) {
    int expert_id = topk_ids[i];
    if (expert_id == -1) {
      continue;
    }
    ++tokens_cnts[(threadIdx.x + 1) * num_experts + expert_id];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    tokens_cnts[threadIdx.x] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[i * num_experts + threadIdx.x] +=
          tokens_cnts[(i - 1) * num_experts + threadIdx.x];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] =
          cumsum[i - 1] +
          CEILDIV(tokens_cnts[blockDim.x * num_experts + i - 1], block_size) *
              block_size;
    }
    *total_tokens_post_pad = static_cast<int32_t>(cumsum[num_experts]);
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    if (expert_id == -1) {
      continue;
    }
    int32_t rank_post_pad =
        tokens_cnts[threadIdx.x * num_experts + expert_id] + cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[threadIdx.x * num_experts + expert_id];
  }
}

template <typename scalar_t>
__global__ void compute_expert_num_tokens_kernel(
    const scalar_t* __restrict__ topk_ids,
    const int32_t* __restrict__ expert_map, int32_t* expert_num_tokens,
    int32_t* sum_expert_num_tokens, const int32_t num_experts,
    const size_t numel) {
  extern __shared__ int32_t shared_mem[];

  for (int x = threadIdx.x; x < num_experts; x += blockDim.x) {
    shared_mem[x] = 0;
  }
  __syncthreads();

  // maybe use register array
  int stride = gridDim.x * blockDim.x;
  int expert_count = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += stride) {
    int const expert_id = topk_ids[i];
    bool valid_expert = expert_id >= 0 && expert_id < num_experts;
    if (expert_map) {
      valid_expert = valid_expert && expert_map[expert_id] != -1;
    }
    if (!valid_expert) {
      continue;
    }
    atomicAdd(&shared_mem[expert_id], 1);
    expert_count++;
  }

  for (int x = threadIdx.x; x < num_experts; x += blockDim.x) {
    atomicAdd(&expert_num_tokens[x], shared_mem[x]);
  }

  atomicAdd(sum_expert_num_tokens, expert_count);
}

}  // namespace moe
}  // namespace vllm

// taken from
// https://github.com/sgl-project/sglang/blob/8b5f83ed3b7d2a49ad5c5cd5aa61c5d502f47dbc
void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts,
                          int64_t block_size, torch::Tensor sorted_token_ids,
                          torch::Tensor experts_ids,
                          torch::Tensor num_tokens_post_pad) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t padded_num_experts =
      ((num_experts + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
  int experts_per_warp = WARP_SIZE;
  int threads = 1024;
  threads = ((threads + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

  VLLM_DISPATCH_INTEGRAL_AND_UNSIGNED_TYPES(
      topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
        // calc needed amount of shared mem for `cumsum` tensors
        auto options_int =
            torch::TensorOptions().dtype(torch::kInt).device(topk_ids.device());
        torch::Tensor cumsum_buffer =
            torch::zeros({num_experts + 1}, options_int);
        bool small_batch_expert_mode =
            (topk_ids.numel() < 1024) && (num_experts <= 64);

        if (small_batch_expert_mode) {
          const int32_t threads = max((int32_t)num_experts, WARP_SIZE);
          const int32_t shared_mem_size =
              ((threads + 1) * num_experts + (num_experts + 1)) *
              sizeof(int32_t);

          auto small_batch_expert_kernel =
              vllm::moe::moe_align_block_size_small_batch_expert_kernel<
                  scalar_t>;
          small_batch_expert_kernel<<<1, threads, shared_mem_size, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts, block_size,
              topk_ids.numel());
        } else {
          auto align_kernel = vllm::moe::moe_align_block_size_kernel<scalar_t>;

          size_t num_warps = CEILDIV(padded_num_experts, experts_per_warp);
          size_t shared_mem_size =
              num_warps * experts_per_warp * sizeof(int32_t);

          align_kernel<<<1, threads, shared_mem_size, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              experts_ids.data_ptr<int32_t>(),
              num_tokens_post_pad.data_ptr<int32_t>(), num_experts,
              padded_num_experts, experts_per_warp, block_size,
              topk_ids.numel(), cumsum_buffer.data_ptr<int32_t>());

          const int block_threads = std::min(256, (int)threads);
          const int num_blocks =
              (topk_ids.numel() + block_threads - 1) / block_threads;
          const int max_blocks = 65535;
          const int actual_blocks = std::min(num_blocks, max_blocks);

          auto sort_kernel =
              vllm::moe::count_and_sort_expert_tokens_kernel<scalar_t>;
          sort_kernel<<<actual_blocks, block_threads, 0, stream>>>(
              topk_ids.data_ptr<scalar_t>(),
              sorted_token_ids.data_ptr<int32_t>(),
              cumsum_buffer.data_ptr<int32_t>(), topk_ids.numel());
        }
      });
}

void moe_sum(torch::Tensor& input,   // [num_tokens, topk, hidden_size]
             torch::Tensor& output)  // [num_tokens, hidden_size]
{
  const int hidden_size = input.size(-1);
  const int num_tokens = output.numel() / hidden_size;
  const int topk = input.size(1);

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  switch (topk) {
    case 2:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 3:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 3><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    case 4:
      VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
        vllm::moe::moe_sum_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
            hidden_size);
      });
      break;

    default:
      at::sum_out(output, input, 1);
      break;
  }
}

void compute_expert_num_tokens(
    torch::Tensor& topk_ids,               // [M, num_topk]
    torch::Tensor& expert_num_tokens,      // [local_num_experts]
    torch::Tensor& sum_expert_num_tokens,  // [1]
    const int32_t local_num_experts,
    std::optional<torch::Tensor> const& expert_map  // [global_num_experts]
) {
  TORCH_CHECK(expert_num_tokens.dtype() == torch::kInt32);
  TORCH_CHECK(sum_expert_num_tokens.dtype() == torch::kInt32);

  if (expert_map) {
    TORCH_CHECK(expert_map->dtype() == torch::kInt32);
  }
  // int device_id = -1;
  // hipGetDevice(&device_id);
  // TORCH_CHECK(device_ids != -1);
  // int num_sms = -1;
  // hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount,
  // device_id); TORCH_CHECK(num_sms != -1);

  // torch::cuda::DeviceProperties properties =
  // torch::cuda::getDeviceProperties(topk_ids.device().index());
  const int num_sms{at::cuda::getDeviceProperties(topk_ids.device().index())
                        ->multiProcessorCount};

  const int topk_numel = topk_ids.numel();
  dim3 block(std::min(topk_numel, 1024));

  const int num_blocks = ((topk_numel - 1 / block.x) + 1) * block.x;
  dim3 grid(std::min(num_sms, num_blocks));

  const at::cuda::OptionalCUDAGuard device_guard(device_of(topk_ids));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  size_t shared_mem_size = local_num_experts * sizeof(int32_t);

  VLLM_DISPATCH_FLOATING_TYPES(
      topk_ids.scalar_type(), "compute_expert_num_tokens_kernel", [&] {
        vllm::moe::compute_expert_num_tokens_kernel<scalar_t>
            <<<grid, block, shared_mem_size, stream>>>(
                topk_ids.data_ptr<scalar_t>(),
                expert_map ? expert_map->data_ptr<int32_t>() : nullptr,
                expert_num_tokens.data_ptr<int32_t>(),
                sum_expert_num_tokens.data_ptr<int32_t>(), local_num_experts,
                topk_numel);
      });
}